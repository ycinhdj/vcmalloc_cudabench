#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <map>
#include <functional>
#include <string>
#include <stdio.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>

#include "cpucounters.h"

extern "C" {
#include "vcmalloc.h"
}

#include "mimalloc.h"

using namespace pcm;

void log_csv(
	const char* operation,
	const char* x,
	const char* y,
	const char* z,
	const char* xtype,
	const char* ytype,
	const char* ztype,
	const char* xunit,
	const char* yunit,
	const char* zunit
) {
	// Print the results
	printf(
		"%s\n"
		"%s (%s): %s\n"
		"%s (%s): %s\n"
		"%s (%s): %s\n",
		operation,
		ztype, zunit, z,
		xtype, xunit, x,
		ytype, yunit, y
	);

	// Save results to a CSV file
	FILE* csv_file = fopen("results.csv", "a");

	// Check if the file is empty; if it is, add headings
	fseek(csv_file, 0, SEEK_END);
	if (ftell(csv_file) == 0) {
		fprintf(csv_file, "operation, x, y, z, xtype, ytype, ztype, xunit, yunit, zunit\n");
	}

	if (csv_file) {
		fprintf(csv_file, "%s, %s, %s, %s, %s, %s, %s, %s, %s, %s\n",
			operation, x, y, z, xtype, ytype, ztype, xunit, yunit, zunit);
		fclose(csv_file);
	}
}

// Define functors for different benchmark types
struct vcm {
	int operator()(int argc, char* argv[]) {

		const char* allocatorname = "vcm";

		PCM* m;
		m = PCM::getInstance();
		m->cleanup();
		PCM::ErrorCode returnResult = m->program();
		if (returnResult != PCM::Success) {
			std::cerr << "PCM couldn't start" << std::endl;
			std::cerr << "Error code: " << returnResult << std::endl;
			exit(1);
		}

		hipError_t cudaStatus;
		clock_t start, end;
		SystemCounterState before_sstate, after_sstate;

		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, 0);

		size_t total_size = prop.totalGlobalMem;
		size_t N = std::strtoull(argv[2], nullptr, 10);
		size_t M = total_size / (N * sizeof(double));

		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		}

		double* dev_buffer;

		cudaStatus = hipMalloc((void**)&dev_buffer, total_size);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
		}

		hca_init(total_size + N * sizeof(double*), N + 1, 1);

		double** buffer = (double**)vca_malloc(N * sizeof(double*));
		for (int i = 0; i < N; ++i)
			buffer[i] = (double*)vca_malloc(M * sizeof(double));

		for (size_t i = 0; i < N; i++)
			for (size_t j = 0; j < M; j++)
				buffer[i][j] = i * M + j;

		start = clock();
		before_sstate = getSystemCounterState();

		for (size_t i = 0; i < N; i++)
		{
			cudaStatus = hipMemcpy(dev_buffer + (i * M), buffer[i], M * sizeof(double), hipMemcpyHostToDevice);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed!");
			}
		}

		after_sstate = getSystemCounterState();
		end = clock();

		double time = (double)(end - start) / CLOCKS_PER_SEC;
		double cpu_energy = getConsumedJoules(before_sstate, after_sstate);
		uint64 L3CacheMisses = getL3CacheMisses(before_sstate, after_sstate);
		uint64 L2CacheMisses = getL2CacheMisses(before_sstate, after_sstate);

		log_csv("CUDA Host to Device",
			std::to_string(N).c_str(), std::to_string(time).c_str(), allocatorname,
			"N", "Time", "Memory Allocator", "", "Seconds", "");

		log_csv("CUDA Host to Device",
			std::to_string(N).c_str(), std::to_string(cpu_energy).c_str(), allocatorname,
			"N", "Energy", "Memory Allocator", "", "Joules", "");

		log_csv("CUDA Host to Device",
			std::to_string(N).c_str(), std::to_string(L3CacheMisses).c_str(), allocatorname,
			"N", "L3 Cache Misses", "Memory Allocator", "", "", "");

		log_csv("CUDA Host to Device",
			std::to_string(N).c_str(), std::to_string(L2CacheMisses).c_str(), allocatorname,
			"N", "L2 Cache Misses", "Memory Allocator", "", "", "");


		start = clock();
		before_sstate = getSystemCounterState();

		for (size_t i = 0; i < N; i++)
		{
			cudaStatus = hipMemcpy(buffer[i], dev_buffer + (i * M), M * sizeof(double), hipMemcpyDeviceToHost);

			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed!");
			}
		}

		after_sstate = getSystemCounterState();
		end = clock();

		time = (double)(end - start) / CLOCKS_PER_SEC;
		cpu_energy = getConsumedJoules(before_sstate, after_sstate);
		L3CacheMisses = getL3CacheMisses(before_sstate, after_sstate);
		L2CacheMisses = getL2CacheMisses(before_sstate, after_sstate);

		log_csv("CUDA Device to Host",
			std::to_string(N).c_str(), std::to_string(time).c_str(), allocatorname,
			"N", "Time", "Memory Allocator", "", "Seconds", "");

		log_csv("CUDA Device to Host",
			std::to_string(N).c_str(), std::to_string(cpu_energy).c_str(), allocatorname,
			"N", "Energy", "Memory Allocator", "", "Joules", "");

		log_csv("CUDA Device to Host",
			std::to_string(N).c_str(), std::to_string(L3CacheMisses).c_str(), allocatorname,
			"N", "L3 Cache Misses", "Memory Allocator", "", "", "");

		log_csv("CUDA Device to Host",
			std::to_string(N).c_str(), std::to_string(L2CacheMisses).c_str(), allocatorname,
			"N", "L2 Cache Misses", "Memory Allocator", "", "", "");

		// hipDeviceReset must be called before exiting in order for profiling and
		// tracing tools such as Nsight and Visual Profiler to show complete traces.
		cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceReset failed!");
			return 1;
		}

		return 0;
	}
};

struct vcma {
	int operator()(int argc, char* argv[]) {

		const char* allocatorname = "vcma";

		PCM* m;
		m = PCM::getInstance();
		m->cleanup();
		PCM::ErrorCode returnResult = m->program();
		if (returnResult != PCM::Success) {
			std::cerr << "PCM couldn't start" << std::endl;
			std::cerr << "Error code: " << returnResult << std::endl;
			exit(1);
		}

		hipError_t cudaStatus;
		clock_t start, end;
		SystemCounterState before_sstate, after_sstate;

		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, 0);

		size_t total_size = prop.totalGlobalMem;
		size_t N = std::strtoull(argv[2], nullptr, 10);
		size_t M = total_size / (N * sizeof(double));

		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		}

		double* dev_buffer;

		cudaStatus = hipMalloc((void**)&dev_buffer, total_size);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
		}


		hca_init(total_size + N * sizeof(double*), N + 1, 1);

		double** buffer = (double**)vca_malloc(N * sizeof(double*));
		for (int i = 0; i < N; ++i)
			buffer[i] = (double*)vca_malloc(M * sizeof(double));

		double* buffer_start = buffer[0];

		for (size_t i = 0; i < N; i++)
			for (size_t j = 0; j < M; j++)
				buffer[i][j] = i * M + j;

		start = clock();
		before_sstate = getSystemCounterState();

		for (size_t i = 0; i < N; i++)
		{
			cudaStatus = hipMemcpy(dev_buffer + (i * M), buffer_start + (i * M), M * sizeof(double), hipMemcpyHostToDevice);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed!");
			}
		}

		after_sstate = getSystemCounterState();
		end = clock();

		double time = (double)(end - start) / CLOCKS_PER_SEC;
		double cpu_energy = getConsumedJoules(before_sstate, after_sstate);
		uint64 L3CacheMisses = getL3CacheMisses(before_sstate, after_sstate);
		uint64 L2CacheMisses = getL2CacheMisses(before_sstate, after_sstate);

		log_csv("CUDA Host to Device",
			std::to_string(N).c_str(), std::to_string(time).c_str(), allocatorname,
			"N", "Time", "Memory Allocator", "", "Seconds", "");

		log_csv("CUDA Host to Device",
			std::to_string(N).c_str(), std::to_string(cpu_energy).c_str(), allocatorname,
			"N", "Energy", "Memory Allocator", "", "Joules", "");

		log_csv("CUDA Host to Device",
			std::to_string(N).c_str(), std::to_string(L3CacheMisses).c_str(), allocatorname,
			"N", "L3 Cache Misses", "Memory Allocator", "", "", "");

		log_csv("CUDA Host to Device",
			std::to_string(N).c_str(), std::to_string(L2CacheMisses).c_str(), allocatorname,
			"N", "L2 Cache Misses", "Memory Allocator", "", "", "");


		start = clock();
		before_sstate = getSystemCounterState();

		for (size_t i = 0; i < N; i++)
		{
			cudaStatus = hipMemcpy(buffer_start + (i * M), dev_buffer + (i * M), M * sizeof(double), hipMemcpyDeviceToHost);

			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed!");
			}
		}

		after_sstate = getSystemCounterState();
		end = clock();

		time = (double)(end - start) / CLOCKS_PER_SEC;
		cpu_energy = getConsumedJoules(before_sstate, after_sstate);
		L3CacheMisses = getL3CacheMisses(before_sstate, after_sstate);
		L2CacheMisses = getL2CacheMisses(before_sstate, after_sstate);

		log_csv("CUDA Device to Host",
			std::to_string(N).c_str(), std::to_string(time).c_str(), allocatorname,
			"N", "Time", "Memory Allocator", "", "Seconds", "");

		log_csv("CUDA Device to Host",
			std::to_string(N).c_str(), std::to_string(cpu_energy).c_str(), allocatorname,
			"N", "Energy", "Memory Allocator", "", "Joules", "");

		log_csv("CUDA Device to Host",
			std::to_string(N).c_str(), std::to_string(L3CacheMisses).c_str(), allocatorname,
			"N", "L3 Cache Misses", "Memory Allocator", "", "", "");

		log_csv("CUDA Device to Host",
			std::to_string(N).c_str(), std::to_string(L2CacheMisses).c_str(), allocatorname,
			"N", "L2 Cache Misses", "Memory Allocator", "", "", "");

		// hipDeviceReset must be called before exiting in order for profiling and
		// tracing tools such as Nsight and Visual Profiler to show complete traces.
		cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceReset failed!");
			return 1;
		}

		return 0;
	}
};

struct vcms {
	int operator()(int argc, char* argv[]) {

		const char* allocatorname = "vcms";

		PCM* m;
		m = PCM::getInstance();
		m->cleanup();
		PCM::ErrorCode returnResult = m->program();
		if (returnResult != PCM::Success) {
			std::cerr << "PCM couldn't start" << std::endl;
			std::cerr << "Error code: " << returnResult << std::endl;
			exit(1);
		}

		hipError_t cudaStatus;
		clock_t start, end;
		SystemCounterState before_sstate, after_sstate;

		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, 0);

		size_t total_size = prop.totalGlobalMem;
		size_t N = std::strtoull(argv[2], nullptr, 10);
		size_t M = total_size / (N * sizeof(double));

		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		}

		double* dev_buffer;

		cudaStatus = hipMalloc((void**)&dev_buffer, total_size);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
		}

		hca_init(total_size + N * sizeof(double*), N + 1, 1);

		double** buffer = (double**)vca_malloc(N * sizeof(double*));
		for (int i = 0; i < N; ++i)
			buffer[i] = (double*)vca_malloc(M * sizeof(double));

		for (size_t i = 0; i < N; i++)
			for (size_t j = 0; j < M; j++)
				buffer[i][j] = i * M + j;

		start = clock();
		before_sstate = getSystemCounterState();

		cudaStatus = hipMemcpy(dev_buffer, buffer[0], N * M * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) fprintf(stderr, "hipMemcpy failed!");

		after_sstate = getSystemCounterState();
		end = clock();

		double time = (double)(end - start) / CLOCKS_PER_SEC;
		double cpu_energy = getConsumedJoules(before_sstate, after_sstate);
		uint64 L3CacheMisses = getL3CacheMisses(before_sstate, after_sstate);
		uint64 L2CacheMisses = getL2CacheMisses(before_sstate, after_sstate);

		log_csv("CUDA Host to Device",
			std::to_string(N).c_str(), std::to_string(time).c_str(), allocatorname,
			"N", "Time", "Memory Allocator", "", "Seconds", "");

		log_csv("CUDA Host to Device",
			std::to_string(N).c_str(), std::to_string(cpu_energy).c_str(), allocatorname,
			"N", "Energy", "Memory Allocator", "", "Joules", "");

		log_csv("CUDA Host to Device",
			std::to_string(N).c_str(), std::to_string(L3CacheMisses).c_str(), allocatorname,
			"N", "L3 Cache Misses", "Memory Allocator", "", "", "");

		log_csv("CUDA Host to Device",
			std::to_string(N).c_str(), std::to_string(L2CacheMisses).c_str(), allocatorname,
			"N", "L2 Cache Misses", "Memory Allocator", "", "", "");


		start = clock();
		before_sstate = getSystemCounterState();

		cudaStatus = hipMemcpy(buffer[0], dev_buffer, N * M * sizeof(double), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) fprintf(stderr, "hipMemcpy failed!");

		after_sstate = getSystemCounterState();
		end = clock();

		time = (double)(end - start) / CLOCKS_PER_SEC;
		cpu_energy = getConsumedJoules(before_sstate, after_sstate);
		L3CacheMisses = getL3CacheMisses(before_sstate, after_sstate);
		L2CacheMisses = getL2CacheMisses(before_sstate, after_sstate);

		log_csv("CUDA Device to Host",
			std::to_string(N).c_str(), std::to_string(time).c_str(), allocatorname,
			"N", "Time", "Memory Allocator", "", "Seconds", "");

		log_csv("CUDA Device to Host",
			std::to_string(N).c_str(), std::to_string(cpu_energy).c_str(), allocatorname,
			"N", "Energy", "Memory Allocator", "", "Joules", "");

		log_csv("CUDA Device to Host",
			std::to_string(N).c_str(), std::to_string(L3CacheMisses).c_str(), allocatorname,
			"N", "L3 Cache Misses", "Memory Allocator", "", "", "");

		log_csv("CUDA Device to Host",
			std::to_string(N).c_str(), std::to_string(L2CacheMisses).c_str(), allocatorname,
			"N", "L2 Cache Misses", "Memory Allocator", "", "", "");

		// hipDeviceReset must be called before exiting in order for profiling and
		// tracing tools such as Nsight and Visual Profiler to show complete traces.
		cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceReset failed!");
			return 1;
		}

		return 0;
	}
};

struct m {
	int operator()(int argc, char* argv[]) {

		const char* allocatorname = "m";

		PCM* m;
		m = PCM::getInstance();
		m->cleanup();
		PCM::ErrorCode returnResult = m->program();
		if (returnResult != PCM::Success) {
			std::cerr << "PCM couldn't start" << std::endl;
			std::cerr << "Error code: " << returnResult << std::endl;
			exit(1);
		}

		hipError_t cudaStatus;
		clock_t start, end;
		SystemCounterState before_sstate, after_sstate;

		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, 0);

		size_t total_size = prop.totalGlobalMem;
		size_t N = std::strtoull(argv[2], nullptr, 10);
		size_t M = total_size / (N * sizeof(double));

		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		}

		double* dev_buffer;

		cudaStatus = hipMalloc((void**)&dev_buffer, total_size);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
		}

		double** buffer = (double**)malloc(N * sizeof(double*));
		for (int i = 0; i < N; ++i)
			buffer[i] = (double*)malloc(M * sizeof(double));



		for (size_t i = 0; i < N; i++)
			for (size_t j = 0; j < M; j++)
				buffer[i][j] = i * M + j;

		start = clock();
		before_sstate = getSystemCounterState();

		for (size_t i = 0; i < N; i++)
		{
			cudaStatus = hipMemcpy(dev_buffer + (i * M), buffer[i], M * sizeof(double), hipMemcpyHostToDevice);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed!");
			}
		}

		after_sstate = getSystemCounterState();
		end = clock();

		double time = (double)(end - start) / CLOCKS_PER_SEC;
		double cpu_energy = getConsumedJoules(before_sstate, after_sstate);
		uint64 L3CacheMisses = getL3CacheMisses(before_sstate, after_sstate);
		uint64 L2CacheMisses = getL2CacheMisses(before_sstate, after_sstate);

		log_csv("CUDA Host to Device",
			std::to_string(N).c_str(), std::to_string(time).c_str(), allocatorname,
			"N", "Time", "Memory Allocator", "", "Seconds", "");

		log_csv("CUDA Host to Device",
			std::to_string(N).c_str(), std::to_string(cpu_energy).c_str(), allocatorname,
			"N", "Energy", "Memory Allocator", "", "Joules", "");

		log_csv("CUDA Host to Device",
			std::to_string(N).c_str(), std::to_string(L3CacheMisses).c_str(), allocatorname,
			"N", "L3 Cache Misses", "Memory Allocator", "", "", "");

		log_csv("CUDA Host to Device",
			std::to_string(N).c_str(), std::to_string(L2CacheMisses).c_str(), allocatorname,
			"N", "L2 Cache Misses", "Memory Allocator", "", "", "");


		start = clock();
		before_sstate = getSystemCounterState();

		for (size_t i = 0; i < N; i++)
		{
			cudaStatus = hipMemcpy(buffer[i], dev_buffer + (i * M), M * sizeof(double), hipMemcpyDeviceToHost);

			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed!");
			}
		}

		after_sstate = getSystemCounterState();
		end = clock();

		time = (double)(end - start) / CLOCKS_PER_SEC;
		cpu_energy = getConsumedJoules(before_sstate, after_sstate);
		L3CacheMisses = getL3CacheMisses(before_sstate, after_sstate);
		L2CacheMisses = getL2CacheMisses(before_sstate, after_sstate);

		log_csv("CUDA Device to Host",
			std::to_string(N).c_str(), std::to_string(time).c_str(), allocatorname,
			"N", "Time", "Memory Allocator", "", "Seconds", "");

		log_csv("CUDA Device to Host",
			std::to_string(N).c_str(), std::to_string(cpu_energy).c_str(), allocatorname,
			"N", "Energy", "Memory Allocator", "", "Joules", "");

		log_csv("CUDA Device to Host",
			std::to_string(N).c_str(), std::to_string(L3CacheMisses).c_str(), allocatorname,
			"N", "L3 Cache Misses", "Memory Allocator", "", "", "");

		log_csv("CUDA Device to Host",
			std::to_string(N).c_str(), std::to_string(L2CacheMisses).c_str(), allocatorname,
			"N", "L2 Cache Misses", "Memory Allocator", "", "", "");

		// hipDeviceReset must be called before exiting in order for profiling and
		// tracing tools such as Nsight and Visual Profiler to show complete traces.
		cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceReset failed!");
			return 1;
		}

		return 0;
	}
};

struct mim {
	int operator()(int argc, char* argv[]) {

		const char* allocatorname = "mim";

		PCM* m;
		m = PCM::getInstance();
		m->cleanup();
		PCM::ErrorCode returnResult = m->program();
		if (returnResult != PCM::Success) {
			std::cerr << "PCM couldn't start" << std::endl;
			std::cerr << "Error code: " << returnResult << std::endl;
			exit(1);
		}

		hipError_t cudaStatus;
		clock_t start, end;
		SystemCounterState before_sstate, after_sstate;

		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, 0);

		size_t total_size = prop.totalGlobalMem;
		size_t N = std::strtoull(argv[2], nullptr, 10);
		size_t M = total_size / (N * sizeof(double));

		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		}

		double* dev_buffer;

		cudaStatus = hipMalloc((void**)&dev_buffer, total_size);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
		}

		double** buffer = (double**)mi_malloc(N * sizeof(double*));
		for (int i = 0; i < N; ++i)
			buffer[i] = (double*)mi_malloc(M * sizeof(double));

		for (size_t i = 0; i < N; i++)
			for (size_t j = 0; j < M; j++)
				buffer[i][j] = i * M + j;

		start = clock();
		before_sstate = getSystemCounterState();

		for (size_t i = 0; i < N; i++)
		{
			cudaStatus = hipMemcpy(dev_buffer + (i * M), buffer[i], M * sizeof(double), hipMemcpyHostToDevice);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed!");
			}
		}

		after_sstate = getSystemCounterState();
		end = clock();

		double time = (double)(end - start) / CLOCKS_PER_SEC;
		double cpu_energy = getConsumedJoules(before_sstate, after_sstate);
		uint64 L3CacheMisses = getL3CacheMisses(before_sstate, after_sstate);
		uint64 L2CacheMisses = getL2CacheMisses(before_sstate, after_sstate);

		log_csv("CUDA Host to Device",
			std::to_string(N).c_str(), std::to_string(time).c_str(), allocatorname,
			"N", "Time", "Memory Allocator", "", "Seconds", "");

		log_csv("CUDA Host to Device",
			std::to_string(N).c_str(), std::to_string(cpu_energy).c_str(), allocatorname,
			"N", "Energy", "Memory Allocator", "", "Joules", "");

		log_csv("CUDA Host to Device",
			std::to_string(N).c_str(), std::to_string(L3CacheMisses).c_str(), allocatorname,
			"N", "L3 Cache Misses", "Memory Allocator", "", "", "");

		log_csv("CUDA Host to Device",
			std::to_string(N).c_str(), std::to_string(L2CacheMisses).c_str(), allocatorname,
			"N", "L2 Cache Misses", "Memory Allocator", "", "", "");


		start = clock();
		before_sstate = getSystemCounterState();

		for (size_t i = 0; i < N; i++)
		{
			cudaStatus = hipMemcpy(buffer[i], dev_buffer + (i * M), M * sizeof(double), hipMemcpyDeviceToHost);

			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed!");
			}
		}

		after_sstate = getSystemCounterState();
		end = clock();

		time = (double)(end - start) / CLOCKS_PER_SEC;
		cpu_energy = getConsumedJoules(before_sstate, after_sstate);
		L3CacheMisses = getL3CacheMisses(before_sstate, after_sstate);
		L2CacheMisses = getL2CacheMisses(before_sstate, after_sstate);

		log_csv("CUDA Device to Host",
			std::to_string(N).c_str(), std::to_string(time).c_str(), allocatorname,
			"N", "Time", "Memory Allocator", "", "Seconds", "");

		log_csv("CUDA Device to Host",
			std::to_string(N).c_str(), std::to_string(cpu_energy).c_str(), allocatorname,
			"N", "Energy", "Memory Allocator", "", "Joules", "");

		log_csv("CUDA Device to Host",
			std::to_string(N).c_str(), std::to_string(L3CacheMisses).c_str(), allocatorname,
			"N", "L3 Cache Misses", "Memory Allocator", "", "", "");

		log_csv("CUDA Device to Host",
			std::to_string(N).c_str(), std::to_string(L2CacheMisses).c_str(), allocatorname,
			"N", "L2 Cache Misses", "Memory Allocator", "", "", "");

		// hipDeviceReset must be called before exiting in order for profiling and
		// tracing tools such as Nsight and Visual Profiler to show complete traces.
		cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceReset failed!");
			return 1;
		}

		return 0;
	}
};

struct ch {
	int operator()(int argc, char* argv[]) {

		const char* allocatorname = "ch";

		PCM* m;
		m = PCM::getInstance();
		m->cleanup();
		PCM::ErrorCode returnResult = m->program();
		if (returnResult != PCM::Success) {
			std::cerr << "PCM couldn't start" << std::endl;
			std::cerr << "Error code: " << returnResult << std::endl;
			exit(1);
		}

		hipError_t cudaStatus;
		clock_t start, end;
		SystemCounterState before_sstate, after_sstate;

		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, 0);

		size_t total_size = prop.totalGlobalMem;
		size_t N = std::strtoull(argv[2], nullptr, 10);
		size_t M = total_size / (N * sizeof(double));

		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		}

		double* dev_buffer;

		cudaStatus = hipMalloc((void**)&dev_buffer, total_size);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
		}

		double** buffer;
		cudaStatus = hipHostAlloc(&buffer, N * sizeof(double*), hipHostMallocDefault);
		if (cudaStatus != hipSuccess) fprintf(stderr, "hipHostAlloc failed!");
		for (int i = 0; i < N; ++i) {
			cudaStatus = hipHostAlloc(&buffer[i], M * sizeof(double), hipHostMallocDefault);
			if (cudaStatus != hipSuccess) fprintf(stderr, "hipHostAlloc failed!");
		}

		for (size_t i = 0; i < N; i++)
			for (size_t j = 0; j < M; j++)
				buffer[i][j] = i * M + j;

		start = clock();
		before_sstate = getSystemCounterState();

		for (size_t i = 0; i < N; i++)
		{
			cudaStatus = hipMemcpy(dev_buffer + (i * M), buffer[i], M * sizeof(double), hipMemcpyHostToDevice);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed!");
			}
		}

		after_sstate = getSystemCounterState();
		end = clock();

		double time = (double)(end - start) / CLOCKS_PER_SEC;
		double cpu_energy = getConsumedJoules(before_sstate, after_sstate);
		uint64 L3CacheMisses = getL3CacheMisses(before_sstate, after_sstate);
		uint64 L2CacheMisses = getL2CacheMisses(before_sstate, after_sstate);

		log_csv("CUDA Host to Device",
			std::to_string(N).c_str(), std::to_string(time).c_str(), allocatorname,
			"N", "Time", "Memory Allocator", "", "Seconds", "");

		log_csv("CUDA Host to Device",
			std::to_string(N).c_str(), std::to_string(cpu_energy).c_str(), allocatorname,
			"N", "Energy", "Memory Allocator", "", "Joules", "");

		log_csv("CUDA Host to Device",
			std::to_string(N).c_str(), std::to_string(L3CacheMisses).c_str(), allocatorname,
			"N", "L3 Cache Misses", "Memory Allocator", "", "", "");

		log_csv("CUDA Host to Device",
			std::to_string(N).c_str(), std::to_string(L2CacheMisses).c_str(), allocatorname,
			"N", "L2 Cache Misses", "Memory Allocator", "", "", "");


		start = clock();
		before_sstate = getSystemCounterState();

		for (size_t i = 0; i < N; i++)
		{
			cudaStatus = hipMemcpy(buffer[i], dev_buffer + (i * M), M * sizeof(double), hipMemcpyDeviceToHost);

			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed!");
			}
		}

		after_sstate = getSystemCounterState();
		end = clock();

		time = (double)(end - start) / CLOCKS_PER_SEC;
		cpu_energy = getConsumedJoules(before_sstate, after_sstate);
		L3CacheMisses = getL3CacheMisses(before_sstate, after_sstate);
		L2CacheMisses = getL2CacheMisses(before_sstate, after_sstate);

		log_csv("CUDA Device to Host",
			std::to_string(N).c_str(), std::to_string(time).c_str(), allocatorname,
			"N", "Time", "Memory Allocator", "", "Seconds", "");

		log_csv("CUDA Device to Host",
			std::to_string(N).c_str(), std::to_string(cpu_energy).c_str(), allocatorname,
			"N", "Energy", "Memory Allocator", "", "Joules", "");

		log_csv("CUDA Device to Host",
			std::to_string(N).c_str(), std::to_string(L3CacheMisses).c_str(), allocatorname,
			"N", "L3 Cache Misses", "Memory Allocator", "", "", "");

		log_csv("CUDA Device to Host",
			std::to_string(N).c_str(), std::to_string(L2CacheMisses).c_str(), allocatorname,
			"N", "L2 Cache Misses", "Memory Allocator", "", "", "");

		// hipDeviceReset must be called before exiting in order for profiling and
		// tracing tools such as Nsight and Visual Profiler to show complete traces.
		cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceReset failed!");
			return 1;
		}

		return 0;
	}
};

struct vcml {
	int operator()(int argc, char* argv[]) {

		const char* allocatorname = "vcml";

		PCM* m;
		m = PCM::getInstance();
		m->cleanup();
		PCM::ErrorCode returnResult = m->program();
		if (returnResult != PCM::Success) {
			std::cerr << "PCM couldn't start" << std::endl;
			std::cerr << "Error code: " << returnResult << std::endl;
			exit(1);
		}

		hipError_t cudaStatus;
		clock_t start, end;
		SystemCounterState before_sstate, after_sstate;

		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, 0);

		size_t total_size = prop.totalGlobalMem;
		size_t N = std::strtoull(argv[2], nullptr, 10);
		size_t M = total_size / (N * sizeof(double));

		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		}

		double* dev_buffer;

		cudaStatus = hipMalloc((void**)&dev_buffer, total_size);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
		}


		hca_init(total_size + N * sizeof(double*), N + 1, 1);

		double** buffer = (double**)vca_malloc(N * sizeof(double*));
		for (int i = 0; i < N; ++i)
			buffer[i] = (double*)vca_malloc(M * sizeof(double));

		//locking memory

		start = clock();
		before_sstate = getSystemCounterState();

		for (int i = 0; i < N; ++i) {
			cudaStatus = hipHostRegister(buffer[i], M * sizeof(double), hipHostRegisterDefault);
			if (cudaStatus != hipSuccess) fprintf(stderr, "hipHostRegister failed!");
		}

		after_sstate = getSystemCounterState();
		end = clock();

		double time = (double)(end - start) / CLOCKS_PER_SEC;
		double cpu_energy = getConsumedJoules(before_sstate, after_sstate);
		uint64 L3CacheMisses = getL3CacheMisses(before_sstate, after_sstate);
		uint64 L2CacheMisses = getL2CacheMisses(before_sstate, after_sstate);

		log_csv("CUDA Host Register",
			std::to_string(N).c_str(), std::to_string(time).c_str(), allocatorname,
			"N", "Time", "Memory Allocator", "", "Seconds", "");

		log_csv("CUDA Host Register",
			std::to_string(N).c_str(), std::to_string(cpu_energy).c_str(), allocatorname,
			"N", "Energy", "Memory Allocator", "", "Joules", "");

		log_csv("CUDA Host Register",
			std::to_string(N).c_str(), std::to_string(L3CacheMisses).c_str(), allocatorname,
			"N", "L3 Cache Misses", "Memory Allocator", "", "", "");

		log_csv("CUDA Host Register",
			std::to_string(N).c_str(), std::to_string(L2CacheMisses).c_str(), allocatorname,
			"N", "L2 Cache Misses", "Memory Allocator", "", "", "");

		for (size_t i = 0; i < N; i++)
			for (size_t j = 0; j < M; j++)
				buffer[i][j] = i * M + j;

		start = clock();
		before_sstate = getSystemCounterState();

		for (size_t i = 0; i < N; i++)
		{
			cudaStatus = hipMemcpy(dev_buffer + (i * M), buffer[i], M * sizeof(double), hipMemcpyHostToDevice);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed!");
			}
		}

		after_sstate = getSystemCounterState();
		end = clock();

		time = (double)(end - start) / CLOCKS_PER_SEC;
		cpu_energy = getConsumedJoules(before_sstate, after_sstate);
		L3CacheMisses = getL3CacheMisses(before_sstate, after_sstate);
		L2CacheMisses = getL2CacheMisses(before_sstate, after_sstate);

		log_csv("CUDA Host to Device",
			std::to_string(N).c_str(), std::to_string(time).c_str(), allocatorname,
			"N", "Time", "Memory Allocator", "", "Seconds", "");

		log_csv("CUDA Host to Device",
			std::to_string(N).c_str(), std::to_string(cpu_energy).c_str(), allocatorname,
			"N", "Energy", "Memory Allocator", "", "Joules", "");

		log_csv("CUDA Host to Device",
			std::to_string(N).c_str(), std::to_string(L3CacheMisses).c_str(), allocatorname,
			"N", "L3 Cache Misses", "Memory Allocator", "", "", "");

		log_csv("CUDA Host to Device",
			std::to_string(N).c_str(), std::to_string(L2CacheMisses).c_str(), allocatorname,
			"N", "L2 Cache Misses", "Memory Allocator", "", "", "");


		start = clock();
		before_sstate = getSystemCounterState();

		for (size_t i = 0; i < N; i++)
		{
			cudaStatus = hipMemcpy(buffer[i], dev_buffer + (i * M), M * sizeof(double), hipMemcpyDeviceToHost);

			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed!");
			}
		}

		after_sstate = getSystemCounterState();
		end = clock();

		time = (double)(end - start) / CLOCKS_PER_SEC;
		cpu_energy = getConsumedJoules(before_sstate, after_sstate);
		L3CacheMisses = getL3CacheMisses(before_sstate, after_sstate);
		L2CacheMisses = getL2CacheMisses(before_sstate, after_sstate);

		log_csv("CUDA Device to Host",
			std::to_string(N).c_str(), std::to_string(time).c_str(), allocatorname,
			"N", "Time", "Memory Allocator", "", "Seconds", "");

		log_csv("CUDA Device to Host",
			std::to_string(N).c_str(), std::to_string(cpu_energy).c_str(), allocatorname,
			"N", "Energy", "Memory Allocator", "", "Joules", "");

		log_csv("CUDA Device to Host",
			std::to_string(N).c_str(), std::to_string(L3CacheMisses).c_str(), allocatorname,
			"N", "L3 Cache Misses", "Memory Allocator", "", "", "");

		log_csv("CUDA Device to Host",
			std::to_string(N).c_str(), std::to_string(L2CacheMisses).c_str(), allocatorname,
			"N", "L2 Cache Misses", "Memory Allocator", "", "", "");

		// hipDeviceReset must be called before exiting in order for profiling and
		// tracing tools such as Nsight and Visual Profiler to show complete traces.
		cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceReset failed!");
			return 1;
		}

		return 0;
	}
};

struct vcmal {
	int operator()(int argc, char* argv[]) {

		const char* allocatorname = "vcmal";

		PCM* m;
		m = PCM::getInstance();
		m->cleanup();
		PCM::ErrorCode returnResult = m->program();
		if (returnResult != PCM::Success) {
			std::cerr << "PCM couldn't start" << std::endl;
			std::cerr << "Error code: " << returnResult << std::endl;
			exit(1);
		}

		hipError_t cudaStatus;
		clock_t start, end;
		SystemCounterState before_sstate, after_sstate;

		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, 0);

		size_t total_size = prop.totalGlobalMem;
		size_t N = std::strtoull(argv[2], nullptr, 10);
		size_t M = total_size / (N * sizeof(double));

		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		}

		double* dev_buffer;

		cudaStatus = hipMalloc((void**)&dev_buffer, total_size);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
		}


		hca_init(total_size + N * sizeof(double*), N + 1, 1);

		double** buffer = (double**)vca_malloc(N * sizeof(double*));
		for (int i = 0; i < N; ++i)
			buffer[i] = (double*)vca_malloc(M * sizeof(double));

		double* buffer_start = buffer[0];

		//locking memory

		start = clock();
		before_sstate = getSystemCounterState();


		for (int i = 0; i < N; ++i) {
			cudaStatus = hipHostRegister(buffer_start + (i * M), M * sizeof(double), hipHostRegisterDefault);
			if (cudaStatus != hipSuccess) fprintf(stderr, "hipHostRegister failed!");
		}

		after_sstate = getSystemCounterState();
		end = clock();

		double time = (double)(end - start) / CLOCKS_PER_SEC;
		double cpu_energy = getConsumedJoules(before_sstate, after_sstate);
		uint64 L3CacheMisses = getL3CacheMisses(before_sstate, after_sstate);
		uint64 L2CacheMisses = getL2CacheMisses(before_sstate, after_sstate);

		log_csv("CUDA Host Register",
			std::to_string(N).c_str(), std::to_string(time).c_str(), allocatorname,
			"N", "Time", "Memory Allocator", "", "Seconds", "");

		log_csv("CUDA Host Register",
			std::to_string(N).c_str(), std::to_string(cpu_energy).c_str(), allocatorname,
			"N", "Energy", "Memory Allocator", "", "Joules", "");

		log_csv("CUDA Host Register",
			std::to_string(N).c_str(), std::to_string(L3CacheMisses).c_str(), allocatorname,
			"N", "L3 Cache Misses", "Memory Allocator", "", "", "");

		log_csv("CUDA Host Register",
			std::to_string(N).c_str(), std::to_string(L2CacheMisses).c_str(), allocatorname,
			"N", "L2 Cache Misses", "Memory Allocator", "", "", "");

		for (size_t i = 0; i < N; i++)
			for (size_t j = 0; j < M; j++)
				buffer[i][j] = i * M + j;

		start = clock();
		before_sstate = getSystemCounterState();

		for (size_t i = 0; i < N; i++)
		{
			cudaStatus = hipMemcpy(dev_buffer + (i * M), buffer_start + (M * i), M * sizeof(double), hipMemcpyHostToDevice);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed!");
			}
		}

		after_sstate = getSystemCounterState();
		end = clock();

		time = (double)(end - start) / CLOCKS_PER_SEC;
		cpu_energy = getConsumedJoules(before_sstate, after_sstate);
		L3CacheMisses = getL3CacheMisses(before_sstate, after_sstate);
		L2CacheMisses = getL2CacheMisses(before_sstate, after_sstate);

		log_csv("CUDA Host to Device",
			std::to_string(N).c_str(), std::to_string(time).c_str(), allocatorname,
			"N", "Time", "Memory Allocator", "", "Seconds", "");

		log_csv("CUDA Host to Device",
			std::to_string(N).c_str(), std::to_string(cpu_energy).c_str(), allocatorname,
			"N", "Energy", "Memory Allocator", "", "Joules", "");

		log_csv("CUDA Host to Device",
			std::to_string(N).c_str(), std::to_string(L3CacheMisses).c_str(), allocatorname,
			"N", "L3 Cache Misses", "Memory Allocator", "", "", "");

		log_csv("CUDA Host to Device",
			std::to_string(N).c_str(), std::to_string(L2CacheMisses).c_str(), allocatorname,
			"N", "L2 Cache Misses", "Memory Allocator", "", "", "");


		start = clock();
		before_sstate = getSystemCounterState();

		for (size_t i = 0; i < N; i++)
		{
			cudaStatus = hipMemcpy(buffer_start + (M * i), dev_buffer + (i * M), M * sizeof(double), hipMemcpyDeviceToHost);

			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed!");
			}
		}

		after_sstate = getSystemCounterState();
		end = clock();

		time = (double)(end - start) / CLOCKS_PER_SEC;
		cpu_energy = getConsumedJoules(before_sstate, after_sstate);
		L3CacheMisses = getL3CacheMisses(before_sstate, after_sstate);
		L2CacheMisses = getL2CacheMisses(before_sstate, after_sstate);

		log_csv("CUDA Device to Host",
			std::to_string(N).c_str(), std::to_string(time).c_str(), allocatorname,
			"N", "Time", "Memory Allocator", "", "Seconds", "");

		log_csv("CUDA Device to Host",
			std::to_string(N).c_str(), std::to_string(cpu_energy).c_str(), allocatorname,
			"N", "Energy", "Memory Allocator", "", "Joules", "");

		log_csv("CUDA Device to Host",
			std::to_string(N).c_str(), std::to_string(L3CacheMisses).c_str(), allocatorname,
			"N", "L3 Cache Misses", "Memory Allocator", "", "", "");

		log_csv("CUDA Device to Host",
			std::to_string(N).c_str(), std::to_string(L2CacheMisses).c_str(), allocatorname,
			"N", "L2 Cache Misses", "Memory Allocator", "", "", "");

		// hipDeviceReset must be called before exiting in order for profiling and
		// tracing tools such as Nsight and Visual Profiler to show complete traces.
		cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceReset failed!");
			return 1;
		}

		return 0;
	}
};

struct vcmsl {
	int operator()(int argc, char* argv[]) {

		const char* allocatorname = "vcmsl";

		PCM* m;
		m = PCM::getInstance();
		m->cleanup();
		PCM::ErrorCode returnResult = m->program();
		if (returnResult != PCM::Success) {
			std::cerr << "PCM couldn't start" << std::endl;
			std::cerr << "Error code: " << returnResult << std::endl;
			exit(1);
		}

		hipError_t cudaStatus;
		clock_t start, end;
		SystemCounterState before_sstate, after_sstate;

		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, 0);

		size_t total_size = prop.totalGlobalMem;
		size_t N = std::strtoull(argv[2], nullptr, 10);
		size_t M = total_size / (N * sizeof(double));

		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		}

		double* dev_buffer;

		cudaStatus = hipMalloc((void**)&dev_buffer, total_size);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
		}


		hca_init(total_size + N * sizeof(double*), N + 1, 1);

		double** buffer = (double**)vca_malloc(N * sizeof(double*));
		for (int i = 0; i < N; ++i)
			buffer[i] = (double*)vca_malloc(M * sizeof(double));


		double* buffer_start = buffer[0];

		//locking memory

		start = clock();
		before_sstate = getSystemCounterState();

		cudaStatus = hipHostRegister(buffer_start, M * N * sizeof(double), hipHostRegisterDefault);
		if (cudaStatus != hipSuccess) fprintf(stderr, "hipHostRegister failed!");

		after_sstate = getSystemCounterState();
		end = clock();

		double time = (double)(end - start) / CLOCKS_PER_SEC;
		double cpu_energy = getConsumedJoules(before_sstate, after_sstate);
		uint64 L3CacheMisses = getL3CacheMisses(before_sstate, after_sstate);
		uint64 L2CacheMisses = getL2CacheMisses(before_sstate, after_sstate);

		log_csv("CUDA Host Register",
			std::to_string(N).c_str(), std::to_string(time).c_str(), allocatorname,
			"N", "Time", "Memory Allocator", "", "Seconds", "");

		log_csv("CUDA Host Register",
			std::to_string(N).c_str(), std::to_string(cpu_energy).c_str(), allocatorname,
			"N", "Energy", "Memory Allocator", "", "Joules", "");

		log_csv("CUDA Host Register",
			std::to_string(N).c_str(), std::to_string(L3CacheMisses).c_str(), allocatorname,
			"N", "L3 Cache Misses", "Memory Allocator", "", "", "");

		log_csv("CUDA Host Register",
			std::to_string(N).c_str(), std::to_string(L2CacheMisses).c_str(), allocatorname,
			"N", "L2 Cache Misses", "Memory Allocator", "", "", "");

		for (size_t i = 0; i < N; i++)
			for (size_t j = 0; j < M; j++)
				buffer[i][j] = i * M + j;

		start = clock();
		before_sstate = getSystemCounterState();

		cudaStatus = hipMemcpy(dev_buffer, buffer[0], N * M * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) fprintf(stderr, "hipMemcpy failed!");

		after_sstate = getSystemCounterState();
		end = clock();

		time = (double)(end - start) / CLOCKS_PER_SEC;
		cpu_energy = getConsumedJoules(before_sstate, after_sstate);
		L3CacheMisses = getL3CacheMisses(before_sstate, after_sstate);
		L2CacheMisses = getL2CacheMisses(before_sstate, after_sstate);

		log_csv("CUDA Host to Device",
			std::to_string(N).c_str(), std::to_string(time).c_str(), allocatorname,
			"N", "Time", "Memory Allocator", "", "Seconds", "");

		log_csv("CUDA Host to Device",
			std::to_string(N).c_str(), std::to_string(cpu_energy).c_str(), allocatorname,
			"N", "Energy", "Memory Allocator", "", "Joules", "");

		log_csv("CUDA Host to Device",
			std::to_string(N).c_str(), std::to_string(L3CacheMisses).c_str(), allocatorname,
			"N", "L3 Cache Misses", "Memory Allocator", "", "", "");

		log_csv("CUDA Host to Device",
			std::to_string(N).c_str(), std::to_string(L2CacheMisses).c_str(), allocatorname,
			"N", "L2 Cache Misses", "Memory Allocator", "", "", "");


		start = clock();
		before_sstate = getSystemCounterState();

		cudaStatus = hipMemcpy(buffer[0], dev_buffer, N * M * sizeof(double), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) fprintf(stderr, "hipMemcpy failed!");

		after_sstate = getSystemCounterState();
		end = clock();

		time = (double)(end - start) / CLOCKS_PER_SEC;
		cpu_energy = getConsumedJoules(before_sstate, after_sstate);
		L3CacheMisses = getL3CacheMisses(before_sstate, after_sstate);
		L2CacheMisses = getL2CacheMisses(before_sstate, after_sstate);

		log_csv("CUDA Device to Host",
			std::to_string(N).c_str(), std::to_string(time).c_str(), allocatorname,
			"N", "Time", "Memory Allocator", "", "Seconds", "");

		log_csv("CUDA Device to Host",
			std::to_string(N).c_str(), std::to_string(cpu_energy).c_str(), allocatorname,
			"N", "Energy", "Memory Allocator", "", "Joules", "");

		log_csv("CUDA Device to Host",
			std::to_string(N).c_str(), std::to_string(L3CacheMisses).c_str(), allocatorname,
			"N", "L3 Cache Misses", "Memory Allocator", "", "", "");

		log_csv("CUDA Device to Host",
			std::to_string(N).c_str(), std::to_string(L2CacheMisses).c_str(), allocatorname,
			"N", "L2 Cache Misses", "Memory Allocator", "", "", "");

		// hipDeviceReset must be called before exiting in order for profiling and
		// tracing tools such as Nsight and Visual Profiler to show complete traces.
		cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceReset failed!");
			return 1;
		}

		return 0;
	}
};

struct ml {
	int operator()(int argc, char* argv[]) {

		const char* allocatorname = "ml";

		PCM* m;
		m = PCM::getInstance();
		m->cleanup();
		PCM::ErrorCode returnResult = m->program();
		if (returnResult != PCM::Success) {
			std::cerr << "PCM couldn't start" << std::endl;
			std::cerr << "Error code: " << returnResult << std::endl;
			exit(1);
		}

		hipError_t cudaStatus;
		clock_t start, end;
		SystemCounterState before_sstate, after_sstate;

		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, 0);

		size_t total_size = prop.totalGlobalMem;
		size_t N = std::strtoull(argv[2], nullptr, 10);
		size_t M = total_size / (N * sizeof(double));

		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		}

		double* dev_buffer;

		cudaStatus = hipMalloc((void**)&dev_buffer, total_size);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
		}


		double** buffer = (double**)malloc(N * sizeof(double*));
		for (int i = 0; i < N; ++i)
			buffer[i] = (double*)malloc(M * sizeof(double));

		//locking memory

		start = clock();
		before_sstate = getSystemCounterState();

		for (int i = 0; i < N; ++i) {
			cudaStatus = hipHostRegister(buffer[i], M * sizeof(double), hipHostRegisterDefault);
			if (cudaStatus != hipSuccess) fprintf(stderr, "hipHostRegister failed!");
		}

		after_sstate = getSystemCounterState();
		end = clock();

		double time = (double)(end - start) / CLOCKS_PER_SEC;
		double cpu_energy = getConsumedJoules(before_sstate, after_sstate);
		uint64 L3CacheMisses = getL3CacheMisses(before_sstate, after_sstate);
		uint64 L2CacheMisses = getL2CacheMisses(before_sstate, after_sstate);

		log_csv("CUDA Host Register",
			std::to_string(N).c_str(), std::to_string(time).c_str(), allocatorname,
			"N", "Time", "Memory Allocator", "", "Seconds", "");

		log_csv("CUDA Host Register",
			std::to_string(N).c_str(), std::to_string(cpu_energy).c_str(), allocatorname,
			"N", "Energy", "Memory Allocator", "", "Joules", "");

		log_csv("CUDA Host Register",
			std::to_string(N).c_str(), std::to_string(L3CacheMisses).c_str(), allocatorname,
			"N", "L3 Cache Misses", "Memory Allocator", "", "", "");

		log_csv("CUDA Host Register",
			std::to_string(N).c_str(), std::to_string(L2CacheMisses).c_str(), allocatorname,
			"N", "L2 Cache Misses", "Memory Allocator", "", "", "");

		for (size_t i = 0; i < N; i++)
			for (size_t j = 0; j < M; j++)
				buffer[i][j] = i * M + j;

		start = clock();
		before_sstate = getSystemCounterState();

		for (size_t i = 0; i < N; i++)
		{
			cudaStatus = hipMemcpy(dev_buffer + (i * M), buffer[i], M * sizeof(double), hipMemcpyHostToDevice);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed!");
			}
		}

		after_sstate = getSystemCounterState();
		end = clock();

		time = (double)(end - start) / CLOCKS_PER_SEC;
		cpu_energy = getConsumedJoules(before_sstate, after_sstate);
		L3CacheMisses = getL3CacheMisses(before_sstate, after_sstate);
		L2CacheMisses = getL2CacheMisses(before_sstate, after_sstate);

		log_csv("CUDA Host to Device",
			std::to_string(N).c_str(), std::to_string(time).c_str(), allocatorname,
			"N", "Time", "Memory Allocator", "", "Seconds", "");

		log_csv("CUDA Host to Device",
			std::to_string(N).c_str(), std::to_string(cpu_energy).c_str(), allocatorname,
			"N", "Energy", "Memory Allocator", "", "Joules", "");

		log_csv("CUDA Host to Device",
			std::to_string(N).c_str(), std::to_string(L3CacheMisses).c_str(), allocatorname,
			"N", "L3 Cache Misses", "Memory Allocator", "", "", "");

		log_csv("CUDA Host to Device",
			std::to_string(N).c_str(), std::to_string(L2CacheMisses).c_str(), allocatorname,
			"N", "L2 Cache Misses", "Memory Allocator", "", "", "");


		start = clock();
		before_sstate = getSystemCounterState();

		for (size_t i = 0; i < N; i++)
		{
			cudaStatus = hipMemcpy(buffer[i], dev_buffer + (i * M), M * sizeof(double), hipMemcpyDeviceToHost);

			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed!");
			}
		}

		after_sstate = getSystemCounterState();
		end = clock();

		time = (double)(end - start) / CLOCKS_PER_SEC;
		cpu_energy = getConsumedJoules(before_sstate, after_sstate);
		L3CacheMisses = getL3CacheMisses(before_sstate, after_sstate);
		L2CacheMisses = getL2CacheMisses(before_sstate, after_sstate);

		log_csv("CUDA Device to Host",
			std::to_string(N).c_str(), std::to_string(time).c_str(), allocatorname,
			"N", "Time", "Memory Allocator", "", "Seconds", "");

		log_csv("CUDA Device to Host",
			std::to_string(N).c_str(), std::to_string(cpu_energy).c_str(), allocatorname,
			"N", "Energy", "Memory Allocator", "", "Joules", "");

		log_csv("CUDA Device to Host",
			std::to_string(N).c_str(), std::to_string(L3CacheMisses).c_str(), allocatorname,
			"N", "L3 Cache Misses", "Memory Allocator", "", "", "");

		log_csv("CUDA Device to Host",
			std::to_string(N).c_str(), std::to_string(L2CacheMisses).c_str(), allocatorname,
			"N", "L2 Cache Misses", "Memory Allocator", "", "", "");

		// hipDeviceReset must be called before exiting in order for profiling and
		// tracing tools such as Nsight and Visual Profiler to show complete traces.
		cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceReset failed!");
			return 1;
		}

		return 0;
	}
};

struct miml {
	int operator()(int argc, char* argv[]) {

		const char* allocatorname = "miml";

		PCM* m;
		m = PCM::getInstance();
		m->cleanup();
		PCM::ErrorCode returnResult = m->program();
		if (returnResult != PCM::Success) {
			std::cerr << "PCM couldn't start" << std::endl;
			std::cerr << "Error code: " << returnResult << std::endl;
			exit(1);
		}

		hipError_t cudaStatus;
		clock_t start, end;
		SystemCounterState before_sstate, after_sstate;

		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, 0);

		size_t total_size = prop.totalGlobalMem;
		size_t N = std::strtoull(argv[2], nullptr, 10);
		size_t M = total_size / (N * sizeof(double));

		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		}

		double* dev_buffer;

		cudaStatus = hipMalloc((void**)&dev_buffer, total_size);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
		}

		double** buffer = (double**)mi_malloc(N * sizeof(double*));
		for (int i = 0; i < N; ++i)
			buffer[i] = (double*)mi_malloc(M * sizeof(double));

		//locking memory

		start = clock();
		before_sstate = getSystemCounterState();

		for (int i = 0; i < N; ++i) {
			cudaStatus = hipHostRegister(buffer[i], M * sizeof(double), hipHostRegisterDefault);
			if (cudaStatus != hipSuccess) fprintf(stderr, "hipHostRegister failed!");
		}

		after_sstate = getSystemCounterState();
		end = clock();

		double time = (double)(end - start) / CLOCKS_PER_SEC;
		double cpu_energy = getConsumedJoules(before_sstate, after_sstate);
		uint64 L3CacheMisses = getL3CacheMisses(before_sstate, after_sstate);
		uint64 L2CacheMisses = getL2CacheMisses(before_sstate, after_sstate);

		log_csv("CUDA Host Register",
			std::to_string(N).c_str(), std::to_string(time).c_str(), allocatorname,
			"N", "Time", "Memory Allocator", "", "Seconds", "");

		log_csv("CUDA Host Register",
			std::to_string(N).c_str(), std::to_string(cpu_energy).c_str(), allocatorname,
			"N", "Energy", "Memory Allocator", "", "Joules", "");

		log_csv("CUDA Host Register",
			std::to_string(N).c_str(), std::to_string(L3CacheMisses).c_str(), allocatorname,
			"N", "L3 Cache Misses", "Memory Allocator", "", "", "");

		log_csv("CUDA Host Register",
			std::to_string(N).c_str(), std::to_string(L2CacheMisses).c_str(), allocatorname,
			"N", "L2 Cache Misses", "Memory Allocator", "", "", "");

		for (size_t i = 0; i < N; i++)
			for (size_t j = 0; j < M; j++)
				buffer[i][j] = i * M + j;

		start = clock();
		before_sstate = getSystemCounterState();

		for (size_t i = 0; i < N; i++)
		{
			cudaStatus = hipMemcpy(dev_buffer + (i * M), buffer[i], M * sizeof(double), hipMemcpyHostToDevice);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed!");
			}
		}

		after_sstate = getSystemCounterState();
		end = clock();

		time = (double)(end - start) / CLOCKS_PER_SEC;
		cpu_energy = getConsumedJoules(before_sstate, after_sstate);
		L3CacheMisses = getL3CacheMisses(before_sstate, after_sstate);
		L2CacheMisses = getL2CacheMisses(before_sstate, after_sstate);

		log_csv("CUDA Host to Device",
			std::to_string(N).c_str(), std::to_string(time).c_str(), allocatorname,
			"N", "Time", "Memory Allocator", "", "Seconds", "");

		log_csv("CUDA Host to Device",
			std::to_string(N).c_str(), std::to_string(cpu_energy).c_str(), allocatorname,
			"N", "Energy", "Memory Allocator", "", "Joules", "");

		log_csv("CUDA Host to Device",
			std::to_string(N).c_str(), std::to_string(L3CacheMisses).c_str(), allocatorname,
			"N", "L3 Cache Misses", "Memory Allocator", "", "", "");

		log_csv("CUDA Host to Device",
			std::to_string(N).c_str(), std::to_string(L2CacheMisses).c_str(), allocatorname,
			"N", "L2 Cache Misses", "Memory Allocator", "", "", "");


		start = clock();
		before_sstate = getSystemCounterState();

		for (size_t i = 0; i < N; i++)
		{
			cudaStatus = hipMemcpy(buffer[i], dev_buffer + (i * M), M * sizeof(double), hipMemcpyDeviceToHost);

			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed!");
			}
		}

		after_sstate = getSystemCounterState();
		end = clock();

		time = (double)(end - start) / CLOCKS_PER_SEC;
		cpu_energy = getConsumedJoules(before_sstate, after_sstate);
		L3CacheMisses = getL3CacheMisses(before_sstate, after_sstate);
		L2CacheMisses = getL2CacheMisses(before_sstate, after_sstate);

		log_csv("CUDA Device to Host",
			std::to_string(N).c_str(), std::to_string(time).c_str(), allocatorname,
			"N", "Time", "Memory Allocator", "", "Seconds", "");

		log_csv("CUDA Device to Host",
			std::to_string(N).c_str(), std::to_string(cpu_energy).c_str(), allocatorname,
			"N", "Energy", "Memory Allocator", "", "Joules", "");

		log_csv("CUDA Device to Host",
			std::to_string(N).c_str(), std::to_string(L3CacheMisses).c_str(), allocatorname,
			"N", "L3 Cache Misses", "Memory Allocator", "", "", "");

		log_csv("CUDA Device to Host",
			std::to_string(N).c_str(), std::to_string(L2CacheMisses).c_str(), allocatorname,
			"N", "L2 Cache Misses", "Memory Allocator", "", "", "");

		// hipDeviceReset must be called before exiting in order for profiling and
		// tracing tools such as Nsight and Visual Profiler to show complete traces.
		cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceReset failed!");
			return 1;
		}

		return 0;
	}
};


std::map<std::string, std::function<int(int argc, char* argv[])>> benchmarkMap = {
	{"vcm", vcm()},
	{"vcma", vcma()},
	{"vcms", vcms()},
	{"vcml", vcml()},
	{"vcmal", vcmal()},
	{"vcmsl", vcmsl()},

	{"m", m()},
	{"ml", ml()},

	{"mim", mim()},
	{"miml", miml()},

	{"ch", ch()},
};

int main(int argc, char* argv[]) {

	if (argc != 3) {
		std::cerr << "Usage: " << argv[0] << " <type> <N>" << std::endl;
		return 1;
	}

	std::string benchmarkType = argv[1];

	auto it = benchmarkMap.find(benchmarkType);
	if (it != benchmarkMap.end()) {
		return it->second(argc, argv);
	}
	else {
		std::cerr << "Invalid benchmark type: " << benchmarkType << std::endl;
		return 1;
	}

}
